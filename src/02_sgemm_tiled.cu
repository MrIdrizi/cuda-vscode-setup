#include "hip/hip_runtime.h"
#include "sgemm.cuh"

// Tiled SGEMM kernel: C = alpha * A @ B + beta * C
__global__ void sgemm_tiled(int M, int N, int K, float alpha, const float *A,
                            const float *B, float beta, float *C) {
    // Block's position inside C
    int b_y = blockIdx.y;
    int b_x = blockIdx.x;

    // Thread's position inside the block (tile)
    int t_y = threadIdx.y;
    int t_x = threadIdx.x;

    // Pointers to the block's top-left (position in C)
    const int C_tile_offs = (BLOCKSIZE * N * b_y) + (BLOCKSIZE * b_x);
    // Offset to row=0 and col=b_x in B
    const int B_tile_offs = BLOCKSIZE * b_x;
    // Offset to row=b_y and col=0 in A
    const int A_tile_offs = (BLOCKSIZE * K) * b_y;

    // Shared-memory buffers for A and B tiles
    __shared__ float As[BLOCKSIZE * BLOCKSIZE];
    __shared__ float Bs[BLOCKSIZE * BLOCKSIZE];

    float tmp = 0.0f;

    // k={0,31,63,...}
    for (int k = 0; k < K; k += BLOCKSIZE) {
        // Load one A value and one B value per thread to shared memory
        if ((k + t_x < K) && (b_y * BLOCKSIZE + t_y < M)) {
            As[BLOCKSIZE * t_y + t_x] = A[A_tile_offs + K * t_y + t_x];
        } else {
            As[BLOCKSIZE * t_y + t_x] = 0.0f; // Out of bounds
        }
        A += BLOCKSIZE;

        if ((b_x * BLOCKSIZE + t_x < N) && (k + t_y < K)) {
            Bs[BLOCKSIZE * t_y + t_x] = B[B_tile_offs + N * t_y + t_x];
        } else {
            Bs[BLOCKSIZE * t_y + t_x] = 0.0f; // Out of bounds
        }
        B += BLOCKSIZE * N;

        // Block threads to wait for all threads to load their values
        __syncthreads();

        // Execute dot product for the current cached block
        for (int i = 0; i < BLOCKSIZE; ++i) {
            tmp += As[BLOCKSIZE * t_y + i] * Bs[BLOCKSIZE * i + t_x];
        }

        __syncthreads();
    }

    if ((b_x * BLOCKSIZE + t_x < N) && (b_y * BLOCKSIZE + t_y < M)) {
        C[C_tile_offs + N * t_y + t_x] =
            alpha * tmp + beta * C[C_tile_offs + N * t_y + t_x];
    }
}
