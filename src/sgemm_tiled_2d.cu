#include "hip/hip_runtime.h"
#include "sgemm.cuh"
#include <cstdio>

// Tiled 2D SGEMM kernel: C = alpha * A @ B + beta * C
__global__ void sgemm_tiled_2d(int M, int N, int K, float alpha, const float *A,
                               const float *B, float beta, float *C) {
    unsigned int bx = blockIdx.x;
    unsigned int by = blockIdx.y;
    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;

    // Pointers to the block's top-left (position in C)
    const int C_tile_offs = N * BM * by + BN * bx;

    // Offset to row=0 and col=bx in B
    int B_tile_offs = BN * bx;
    // Offset to row=by and col=0 in A
    int A_tile_offs = BM * K * by;

    // C += beta * C
    // This could also be done later but I use C to store intermediate results
    for (int tm = 0; tm < TM; ++tm) {
        for (int tn = 0; tn < TN; ++tn) {
            C[C_tile_offs + N * (TM * ty + tm) + TN * tx + tn] *= beta;
        }
    }

    // Shared-memory buffers for A and B tiles
    __shared__ float As[BM * BK];
    __shared__ float Bs[BK * BN];

    // k = {0, BK, 2*BK, ...}
    for (int k = 0; k < K; k += BK) {
        // Each thread loads TM values into As
        for (int tm = 0; tm < TM; ++tm) {
            As[BK * (TM * ty + tm) + tx] =
                A[A_tile_offs + K * (TM * ty + tm) + tx];
        }
        A_tile_offs += BK;

        // Each thread loads TN values into Bs
        for (int tn = 0; tn < TN; ++tn) {
            Bs[BN * ty + TN * tx + tn] = B[B_tile_offs + N * ty + TN * tx + tn];
        }
        B_tile_offs += N * BK;
        __syncthreads();

        // Each thread computes a TMxTN block
        float tmp[TM][TN] = {0.0f};
        for (int tm = 0; tm < TM; ++tm) {
            for (int tn = 0; tn < TN; ++tn) {
                for (int bk = 0; bk < BK; ++bk) {
                    tmp[tm][tn] += As[BK * (TM * ty + tm) + bk] *
                                   Bs[TN * tx + tn + BN * bk];
                }
            }
        }

        // Each thread copies its part of the block to C
        for (int tm = 0; tm < TM; ++tm) {
            for (int tn = 0; tn < TN; ++tn) {
                C[C_tile_offs + N * (TM * ty + tm) + TN * tx + tn] =
                    alpha * tmp[tm][tn] +
                    C[C_tile_offs + N * (TM * ty + tm) + TN * tx + tn];
            }
        }
        __syncthreads();
    }
}
